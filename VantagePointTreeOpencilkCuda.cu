#include "hip/hip_runtime.h"
#include <cilk/cilk.h>
#include <iostream>
#include <vector>
#include <algorithm>
#include <cmath>
#include <ctime>
#include <chrono>
#include <tbb/mutex.h> //mutex library
using namespace std;

using Points = vector<double>;

const int THREADS_PER_BLOCK = 256;

struct Node{
    Node(const vector<double>& vantagePoint, double medianDistance, unsigned int vantagePointIndex) : vantagePoint(vantagePoint), medianDistance(medianDistance), vantagePointIndex(vantagePointIndex), inner(nullptr), outer(nullptr) {}

    vector<double> vantagePoint;
    double medianDistance;
    unsigned int vantagePointIndex;
    Node *inner, *outer;
};

struct Tree{
    Node* root;
};

vector<double> GetDistancesCPU(const Points& points, const vector<double>& vantagePoint){
    unsigned int dimensionsTotal = vantagePoint.size();
    unsigned int pointsTotal = points.size() / dimensionsTotal;

    vector<double> distances(pointsTotal - 1);

    for(int i = 0; i < pointsTotal - 1; i++){
        double squareDistance;squareDistance = 0;
        for(int j = i * dimensionsTotal; j < (i + 1) * dimensionsTotal; j++)
            squareDistance += pow(vantagePoint[j - (i * dimensionsTotal)] - points[j], 2);
        double distance = sqrt(squareDistance);
        distances[i] = distance;
    }

    return distances;
}

__global__ void GetDistancesKernel(double* points, double* vantagePoint, double* distances, int pointsTotal, int dimensionsTotal){
    int pointIndex = threadIdx.x + blockIdx.x * blockDim.x;

    if(pointIndex < pointsTotal){
        double squareDistance = 0;
        int startingIndex = pointIndex * dimensionsTotal;
        for(int i = startingIndex; i < startingIndex + dimensionsTotal; i++)
            squareDistance += pow(vantagePoint[i - startingIndex] - points[i], 2);
        double distance = sqrt(squareDistance);
        distances[pointIndex] = distance;
    }
}

tbb::mutex gpuMutex;

vector<double> GetDistancesGPU(const Points& points, const vector<double>& vantagePoint){
    static unsigned int kernelsCalled = 0;

    static double* vantagePointDevicePointer; // device pointers
    static double* distancesDevicePointer; 
    static double* pointsDevicePointer;

    unsigned int dimensionsTotal = vantagePoint.size();
    unsigned int pointsTotal = points.size() / dimensionsTotal;

    vector<double> distances(pointsTotal - 1);

    const double* pointsHostPointer = points.data(); // host pointers
    const double* vantagePointHostPointer = vantagePoint.data(); 
    double* distancesHostPointer = distances.data();

    if(kernelsCalled == 0){
        kernelsCalled = 1; // when this line will be executed we will still have just one thread in the program so no race conditions can occur
        hipMalloc(&pointsDevicePointer, ((pointsTotal - 1) * dimensionsTotal) * sizeof(double)); // allocate device space
        hipMalloc(&distancesDevicePointer, (pointsTotal - 1) * sizeof(double));
        hipMalloc(&vantagePointDevicePointer, dimensionsTotal * sizeof(double));
    }

    
    gpuMutex.lock(); // the next 4 instructions should be executed by only one cpu thread at a time so that no gpu memory is overwritten 

    hipMemcpy(pointsDevicePointer, pointsHostPointer, ((pointsTotal - 1) * dimensionsTotal) * sizeof(double), hipMemcpyHostToDevice); // copy data from host to device memory 
    hipMemcpy(vantagePointDevicePointer, vantagePointHostPointer, dimensionsTotal * sizeof(double), hipMemcpyHostToDevice);

    
    GetDistancesKernel<<<(pointsTotal + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(pointsDevicePointer, vantagePointDevicePointer, distancesDevicePointer, pointsTotal - 1, dimensionsTotal); // kernel call


    hipMemcpy(distancesHostPointer, distancesDevicePointer, (pointsTotal - 1) * sizeof(double), hipMemcpyDeviceToHost); // copy data from device to host memory

    gpuMutex.unlock();

    return distances;
}

double GetMedianDistance(vector<double> distances){
    unsigned int distancesTotal = distances.size();
    if(distancesTotal % 2 == 0){
        nth_element(distances.begin(), distances.begin() + distancesTotal/2, distances.end());
        nth_element(distances.begin(), distances.begin() + (distancesTotal-1)/2, distances.end());
        return (distances[(distancesTotal-1)/2] + distances[distancesTotal/2]) / 2;
    }
    else{
        nth_element(distances.begin(), distances.begin() + distancesTotal/2, distances.end());
        return distances[distancesTotal/2];
    }
}

void VantageTreeConstructionHelper(Node*& node, const Points& points, const vector<unsigned int> indices, unsigned int dimensionsTotal, unsigned int neighborsTotal, unsigned int neighborsCurrent = 0){
    unsigned int pointsTotal = points.size() / dimensionsTotal;
    
    if(neighborsCurrent == neighborsTotal || pointsTotal <= 1) 
        return;
    
    vector<double> vantagePoint(points.end() - dimensionsTotal, points.end());
    unsigned int vantagePointIndex = indices[pointsTotal - 1];

    vector<double> distances;
    if(pointsTotal > 1000)
        distances = GetDistancesGPU(points, vantagePoint);    
    else 
        distances = GetDistancesCPU(points, vantagePoint);
        
    double medianDistance = GetMedianDistance(distances);

    node = new Node(vantagePoint, medianDistance, vantagePointIndex);

    Points innerPoints, outerPoints;
    vector<unsigned int> innerPointsIndices, outerPointsIndices;
    for(int i = 0; i < pointsTotal - 1; i++){
        if(distances[i] < medianDistance){
            for(int j = i * dimensionsTotal; j < (i + 1) * dimensionsTotal; j++)
                innerPoints.push_back(points[j]);
            innerPointsIndices.push_back(indices[i]);
        }
        else{
            for(int j = i * dimensionsTotal; j < (i + 1) * dimensionsTotal; j++)
                outerPoints.push_back(points[j]);
            outerPointsIndices.push_back(indices[i]);
        }
    } 

    if(innerPoints.size() > 32)
        cilk_spawn VantageTreeConstructionHelper(node->inner, innerPoints, innerPointsIndices, dimensionsTotal, neighborsTotal, neighborsCurrent + 1);
    else VantageTreeConstructionHelper(node->inner, innerPoints, innerPointsIndices, dimensionsTotal, neighborsTotal, neighborsCurrent + 1);

    VantageTreeConstructionHelper(node->outer, outerPoints, outerPointsIndices, dimensionsTotal, neighborsTotal, neighborsCurrent + 1);

    cilk_sync;
}

vector<unsigned int> GetInitialIndices(const Points& points, unsigned int dimensionsTotal){
    unsigned int pointsTotal = points.size() / dimensionsTotal;
    vector<unsigned int> indices(pointsTotal);

    for(int i = 0; i < pointsTotal; i++){
        indices[i] = i;
    }

    return indices;
}

Tree VantageTreeConstruction(const Points& points, unsigned int neighborsTotal, unsigned int dimensionsTotal){
    Tree vantagePointTree;

    vector<unsigned int> indices = GetInitialIndices(points, dimensionsTotal);
    VantageTreeConstructionHelper(vantagePointTree.root, points, indices, dimensionsTotal, neighborsTotal);

    return vantagePointTree;
}

Points generateRandomPoints(unsigned int pointsTotal, unsigned int dimensionsTotal){
    Points points(pointsTotal*dimensionsTotal);
    
    for(int i = 0; i < pointsTotal*dimensionsTotal; i++)
        points[i] = rand();

    return points;
}

int main(){
    srand(time_t(time(NULL)));

    unsigned int pointsTotal = 1000000;
    unsigned int dimensionsTotal = 3;

    Points points = generateRandomPoints(pointsTotal, dimensionsTotal);

    unsigned int neighborsTotal = 7;

    chrono::high_resolution_clock::time_point start, end;

    start = chrono::high_resolution_clock::now();
    Tree vantagePointTree = VantageTreeConstruction(points, neighborsTotal, dimensionsTotal);
    end = chrono::high_resolution_clock::now();

    cout << chrono::duration_cast<chrono::duration<double>>(end - start).count() << endl;
}